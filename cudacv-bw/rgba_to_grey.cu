#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  uchar4 px = rgbaImage[i]; // thread pixel to process
  greyImage[i] = .299f * px.x +
                 .587f * px.y +
                 .114f * px.z;
    
  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(512, 1, 1);  //TODO
  const dim3 gridSize(512, 1, 1);  //TODO
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}